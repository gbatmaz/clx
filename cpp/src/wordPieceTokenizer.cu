#include "hip/hip_runtime.h"
#include <limits>
#include <stdint.h>

#include "data_transfer_utils.cuh"
#include "hipcub/hipcub.hpp"
#include "tokenizer_utils.cuh"
#include "hash_utils.cuh"
#include "tokenizers.cuh"

__device__ __forceinline__ void __init_data_and_mark_word_start_and_ends(uint32_t* code_points, uint32_t* start_word_indices, 
                                                                         uint32_t* end_word_indices, size_t num_code_points,
                                                                         uint32_t* token_ids, uint8_t* tokens_per_word,
                                                                         uint32_t char_for_thread) {
  // Deal with the start_word_indices array
  if(char_for_thread < num_code_points) { 
    uint32_t val_to_write = std::numeric_limits<uint32_t>::max();
    if((code_points[char_for_thread] != SPACE_CODE_POINT) && (char_for_thread > 0) && (code_points[char_for_thread - 1] == SPACE_CODE_POINT)) {
      val_to_write = char_for_thread;
    }
    start_word_indices[char_for_thread] = val_to_write;

    // Deal with the end_word_indices_aray
    val_to_write = std::numeric_limits<uint32_t>::max();
    if((code_points[char_for_thread] != SPACE_CODE_POINT) && (char_for_thread + 1 < num_code_points) && 
        (code_points[char_for_thread + 1] == SPACE_CODE_POINT)) {
      val_to_write = char_for_thread + 1;
    }
    end_word_indices[char_for_thread] = val_to_write;

    token_ids[char_for_thread] = std::numeric_limits<uint32_t>::max();
    tokens_per_word[char_for_thread] = 0;
  }
}

__device__ __forceinline__ void __mark_sentence_start_and_ends(uint32_t* code_points, uint32_t* sentence_offsets,
                                                               uint32_t* start_word_indices, uint32_t* end_word_indices,
                                                               uint32_t num_sentences, uint32_t char_for_thread) {

  // Ensure the starting character of each sentence is written to the word start array.
  if(char_for_thread <= num_sentences) {
    const uint32_t offset = sentence_offsets[char_for_thread];

    if((char_for_thread < num_sentences) && (code_points[offset] != SPACE_CODE_POINT)) {
      start_word_indices[offset] = offset;
    }

    if((char_for_thread > 0) && (code_points[offset - 1] != SPACE_CODE_POINT)) {
      end_word_indices[offset - 1] = offset;
    }
  }
}

/*
  Writes the index to each thread which points to the start of a word to idx_for_sen_start.

  Params
  -------
  code_points: A pointer to the code points in the sentence after being run through the basic
               GPU tokenizer.

  start_word_indices: An array which will contain the starting index for each word scattered throughout.
                      If an index does not represent a word start, the max uint32_t value is written 
                      to indicate this. A post processing step is required to select all the relevant 
                      values from this array.
  
  end_word_indices: An array which will contain the one past the end index for each word scattered throughout.
                    If an index does not represent a word end, the max uint32_t value is written 
                    to indicate this. A post processing step is required to select all the relevant 
                    values from this array. 

                    It is guaranteed that the same number of indices will be written to each kernel 
                    and that after the select step, the two arrays will be aligned (ie. 
                    start_word_indices[word] and end_word_indices[word] are the start and
                    end for the same word). This is not true before the hipcub::deviceselect is done.

  num_code_points: The total number of code_points in the code_points array.

  token_ids: The array which will hold the token ids. This kernel initialized all values in this array to
             the max uint32_t. It is assumed that the length of this array is num_code_points.
  
  tokens_per_word: The array which will hold the number of tokens in each word. This kernel initialized all
                   values in this array to 0. It is assumed that the length of this array is num_code_points.
*/
__global__ void init_data_and_mark_word_start_and_ends(uint32_t* code_points, uint32_t* start_word_indices, 
                                                       uint32_t* end_word_indices, size_t num_code_points,
                                                       uint32_t* token_ids, uint8_t* tokens_per_word) {

  uint32_t char_for_thread = blockDim.x * blockIdx.x + threadIdx.x;

  __init_data_and_mark_word_start_and_ends(code_points, start_word_indices, end_word_indices, num_code_points,
                                            token_ids, tokens_per_word, char_for_thread);
}


/*
  Writes the indicies of the characters that start sentences in the start_word_indices array and index 
  of the character after the last character in the sentence to the end_word_indices array. This kernel
  should be called after mark_word_start_and_ends with at least num_sentences total threads.

  Params
  -------
  code_points: A pointer to the code points in the sentence after being run through the basic
               GPU tokenizer.
  
  sentence_offsets: an array containing the index of the starting character of each sentence with
                    an extra space at the end containing the total number of characters. As a result,
                    this array is of length num_sentences + 1.

  start_word_indices: An array which will contain the starting index for each word scattered throughout.
                      If an index does not represent a word start, the max uint32_t value is written 
                      to indicate this. A post processing step is required to select all the relevant 
                      values from this array.
  
  end_word_indices: An array which will contain the one past the end index for each word scattered throughout.
                    If an index does not represent a word end, the max uint32_t value is written 
                    to indicate this. A post processing step is required to select all the relevant 
                    values from this array. 

                    It is guaranteed that the same number of indices will be written to each kernel 
                    and that after the select step, the two arrays will be aligned (ie. 
                    start_word_indices[word] and end_word_indices[word] are the start and
                    end for the same word). This is not true before the hipcub::deviceselect is done.

  num_sentences: The total number of sentences to be processed.
*/
__global__ void mark_sentence_start_and_ends(uint32_t* code_points, uint32_t* sentence_offsets,
                                             uint32_t* start_word_indices, uint32_t* end_word_indices,
                                             uint32_t num_sentences) {

  uint32_t char_for_thread = blockDim.x * blockIdx.x + threadIdx.x;
  __mark_sentence_start_and_ends(code_points, sentence_offsets, start_word_indices, end_word_indices,
                                 num_sentences, char_for_thread);
}

/* 
  A helper function for gpuWordPieceTokenizer.
  See the spec of gpuWordPieceTokenizer for parameter details. This function 
  takes token_start and token_end as inputs which is the start and end indices 
  for each token in the code_points array.
*/
__device__ __forceinline__ void __wordPieceTokenize(uint32_t* code_points, uint64_t* hash_table, uint64_t* bin_coefficients, 
                                                    uint16_t* bin_offsets, uint32_t* token_ids, const uint32_t token_start, 
                                                    const uint32_t token_end, uint8_t* tokens_per_word, uint16_t  unk_token_id, 
                                                    uint16_t max_word_length, uint32_t outer_hash_a_param, uint32_t outer_hash_b_param, 
                                                    uint16_t num_outer_bins) {

  // The sdbm hash of "##"
  constexpr uint32_t hashtag_hash = 2296000;

  uint32_t end = token_end, start = token_start; 
  const uint32_t word_length = token_end - token_start;  
  uint16_t num_values_tokenized = 0;
                                                   
  if(word_length > max_word_length) {
    start = token_end;
    num_values_tokenized = 1;
    token_ids[token_start] = unk_token_id;
    tokens_per_word[token_start] = num_values_tokenized;
  }

  while(start < token_end) {
    end = token_end;
    int token_id = -1;
    const uint32_t length = token_end - start;
    uint64_t substr_hash = sdbm_hash(code_points + start, length, start == token_start? 0: hashtag_hash);

    while(start < end) {
      token_id = retrieve(substr_hash, outer_hash_a_param, outer_hash_b_param, num_outer_bins, hash_table, bin_coefficients, bin_offsets);
      if(token_id != -1) {
        break;
      }
      --end;
      // Pop off the last value from the substr hash
      substr_hash = prev_sdbm_hash(substr_hash, code_points[end]);
    }

    if(token_id == -1) {
      end = token_end;
      token_id = unk_token_id;

      // We need to clean up the global array. This case is very uncommon. Only 0.016% of words cannot be
      // resolved to a token from the squad dev set.
      for(uint32_t i = 1; i < num_values_tokenized; ++i) {
        token_ids[token_start + i] = std::numeric_limits<uint32_t>::max();
      }

      num_values_tokenized = 0;
    }

    token_ids[token_start + num_values_tokenized] = token_id;
    ++num_values_tokenized;
    start = end;
  }
  
  tokens_per_word[token_start] = num_values_tokenized;
}

/*
  Splits words into their token ids. 

  Some implementation details:

  Each thread is assigned a word to tokenize based on thread_to_word_map. Each thread tokenizes
  its word and writes the number of tokens it found in the tokens_per_word array. 

  The tokens_per_word array is kept to the length (num_code_points + 1). This means each thread
  can write its number of tokens to the index in thread_to_word_map corresponding to the starting
  character of each word. Since sentences must start at some word, we can prefix sum this array 
  and use the sentence_lengths code point offsets to directly index the number of tokens in each
  sentence.

  Params:
  code_points: an array containing all of the code points to be processed

  hash_table: An array containing the flattened hash table with key, value pairs packed in 64-bits

  device_bin_coefficients: A pointer to the GPU pointer containing the hashing parameters for
                           each hash bin on the GPU.
  
  device_bin_offsets: A pointer to the GPU pointer containing the start index of each bin in 
                      the flattened hash table.

  token_ids: The index for each token found during tokenization. This is of length num_code_points. 
             In most cases, multiple characters will collapse to one token. In these cases, the max
             uint32_t will be in place. Cub will be used later to filter out these invalid ids later.

             This array should be initialized to the max uint32_t before calling this kernel.

  word_starts: An array of length num_code_points. The first total word elements contains the index
               of the first character for each word.              

  word_ends: An array of length num_code_points. The first total_words elements contains the 
             past the end index for each word. This array is kept aligned with the initial token_ids
             array containing the word start code points. Thus, word_ends[word] - filtered_start_indices[word] = word_length          
  
  tokens_per_word: An array of size num_code_points that will contain the number of tokens in each 
                   word in a sentence. 
                   This array can be exclusive summed and the result used in conjunction with the sentence 
                   lengths array to find the tokens in each sentence. This is possible since the number of
                   tokens in each word will be placed at the index corresponding to the start character of 
                   a word. 
                   If we assume prefix_summed is the prefix sum of the tokens_per_word array, then 
                   prefix_summed[sentence_lengths[sentence] - 1] is the number of tokens found before the
                   start of sentence. 

  unk_token_id: The token id to be place for unknown tokens

  max_word_length: The maximum length of a word. Any word longer than this length is replaced by the unknown
                   token.
    
  total_words: The total number of white space separated words

  outer_hash_a_param: The a parameter for the outer hash

  outer_hash_b_param: The b parameter for the outer hash

  num_outer_bins: The number of bins for the outer hash
*/
__global__ void gpuWordPieceTokenizer(uint32_t* code_points, uint64_t* hash_table, uint64_t* bin_coefficients, 
                                      uint16_t* bin_offsets, uint32_t* token_ids, uint32_t* word_starts, 
                                      uint32_t* word_ends, uint8_t* tokens_per_word, uint16_t  unk_token_id, 
                                      uint16_t max_word_length, uint32_t total_words, uint32_t outer_hash_a_param, 
                                      uint32_t outer_hash_b_param, uint16_t num_outer_bins) {

  const uint32_t word_to_tokenize = blockDim.x * blockIdx.x + threadIdx.x;

  if(word_to_tokenize < total_words) {

    // Each thread gets the start code_point offset for each word and resets the token_id memory to
    // the default value. In a post processing step, all of these values will be removed.
    const uint32_t token_start = word_starts[word_to_tokenize];
    const uint32_t token_end = word_ends[word_to_tokenize];

    __wordPieceTokenize(code_points, hash_table, bin_coefficients, bin_offsets, token_ids, token_start, 
                        token_end, tokens_per_word, unk_token_id, max_word_length, outer_hash_a_param, outer_hash_b_param, 
                        num_outer_bins);
  }
}

// ---------------------------------------- Word Piece tokenizer definitions ------------------------------------------------------
// See tokenizers.cuh
GpuWordPieceTokenizer::GpuWordPieceTokenizer(std::string vocab_file, uint32_t max_num_chars, uint32_t max_inp_chars_per_word): 
device_token_ids{},
device_word_indices{},
device_tokens_per_word{},
device_hash_table{},
device_bin_coefficients{},
device_bin_offsets{} {

  transfer_hash_info_to_device(vocab_file, device_hash_table, device_bin_coefficients, device_bin_offsets,
                               unk_token_id, first_tok_id, sep_tok_id, outer_hash_a_param, outer_hash_b_param,
                               num_outer_bins);

  max_word_length = max_inp_chars_per_word;
  
  const size_t max_new_char_total = MAX_NEW_CHARS * max_num_chars;
  device_token_ids.resize(max_new_char_total);
  const size_t device_word_indices_count = 2 * max_new_char_total;
  device_word_indices.resize(device_word_indices_count);

  const size_t four_byte_cp_chunks = 1 + (max_new_char_total - 1) / sizeof(uint32_t);
  const size_t rounded_num_cps = sizeof(uint32_t) * four_byte_cp_chunks;
  device_tokens_per_word.resize(rounded_num_cps);

  // Determine temporary device storage requirements for cub
  static NotEqual select_op(std::numeric_limits<uint32_t>::max());
  size_t temp_storage_bytes = 0, temp_storage_bytes_2 = 0;
  hipcub::DeviceSelect::If(nullptr, temp_storage_bytes, thrust::raw_pointer_cast(device_word_indices.data()), thrust::raw_pointer_cast(device_word_indices.data()), 
                        thrust::raw_pointer_cast(device_num_selected.data()), 2*max_new_char_total, select_op);
  hipcub::DeviceScan::InclusiveSum(nullptr, temp_storage_bytes_2, thrust::raw_pointer_cast(device_tokens_per_word.data()), 
                        thrust::raw_pointer_cast(device_word_indices.data()), max_new_char_total);
  max_cub_storage_bytes = std::max(temp_storage_bytes, temp_storage_bytes_2);
  cub_temp_storage.resize(max_cub_storage_bytes);
  device_num_selected.resize(1);  
 }



void GpuWordPieceTokenizer::tokenize(ptr_length_pair<uint32_t*>& cp_and_length, 
                                     ptr_length_pair<uint32_t*>& offsets_and_length) {

  uint32_t* device_code_points = cp_and_length.gpu_ptr;
  size_t num_code_points = cp_and_length.length;

  uint32_t* device_sentence_offsets = offsets_and_length.gpu_ptr;
  uint32_t num_sentences = offsets_and_length.length - 1;

  // Create a selection op for all device selects                                                    
  static NotEqual select_op(std::numeric_limits<uint32_t>::max());

  // make device_start_word_indices and device_end_word_indices contiguous
  uint32_t* device_start_word_indices = thrust::raw_pointer_cast(device_word_indices.data());
  uint32_t* device_end_word_indices = device_start_word_indices + num_code_points;
  
  uint32_t total_threads = num_code_points;
  constexpr uint32_t threads_per_block = 64;
  uint32_t num_blocks = (total_threads + threads_per_block - 1) / threads_per_block;  
  init_data_and_mark_word_start_and_ends<<<num_blocks, threads_per_block>>>(device_code_points, device_start_word_indices, device_end_word_indices, 
                                                                            num_code_points, thrust::raw_pointer_cast(device_token_ids.data()), thrust::raw_pointer_cast(device_tokens_per_word.data()));
  assertCudaSuccess(hipPeekAtLastError());  

  uint32_t word_split_blocks = (num_sentences + threads_per_block - 1) / threads_per_block;                                                              
  mark_sentence_start_and_ends<<<word_split_blocks, threads_per_block>>>(device_code_points, device_sentence_offsets, device_start_word_indices, 
                                                                         device_end_word_indices, num_sentences);
  assertCudaSuccess(hipPeekAtLastError());  

  // Now start_word_indices has the word starts scattered throughout the array. We need to select all values not equal to the max uint32_t 
  // and place them at the start of the array. We leverage the fact that the start_word_indices and the end_word indices are contiguous to
  // only launch one device select kernel.
  hipcub::DeviceSelect::If(thrust::raw_pointer_cast(cub_temp_storage.data()), max_cub_storage_bytes, device_start_word_indices, device_start_word_indices, thrust::raw_pointer_cast(device_num_selected.data()), 2*num_code_points, select_op);
  assertCudaSuccess(hipPeekAtLastError());  

  // Grab the number of words which is the number of threads needed for the main word piece tokenizer kernel. The number of tokens selected out will
  // be double the number of words since we select from both the start and end index arrays.
  uint32_t num_words = 0;
  device_num_selected.resize(1);
  assertCudaSuccess(hipMemcpy(&num_words, thrust::raw_pointer_cast(device_num_selected.data()), sizeof(num_words), hipMemcpyDeviceToHost));
  
  num_words /= 2;

  // We need to change the end_word_indices pointer after the selection is complete
  device_end_word_indices = device_start_word_indices + num_words;
    
  const uint32_t wp_threads_per_block = 64;
  const uint32_t num_wp_blocks = (num_words + wp_threads_per_block - 1) / wp_threads_per_block;
  gpuWordPieceTokenizer<<<num_wp_blocks, wp_threads_per_block>>>(device_code_points, thrust::raw_pointer_cast(device_hash_table.data()), thrust::raw_pointer_cast(device_bin_coefficients.data()), thrust::raw_pointer_cast(device_bin_offsets.data()), 
    thrust::raw_pointer_cast(device_token_ids.data()), device_start_word_indices, device_end_word_indices, thrust::raw_pointer_cast(device_tokens_per_word.data()), 
                                                                 unk_token_id, max_word_length, num_words, outer_hash_a_param, outer_hash_b_param, num_outer_bins);
  assertCudaSuccess(hipPeekAtLastError());  
  
  // Repurpose the input array for the token ids. In the worst case, each code point ends up being a token so this will
  // always have enough memory to store the contiguous tokens.
  uint32_t* contiguous_token_ids = device_code_points;
  hipcub::DeviceSelect::If(thrust::raw_pointer_cast(cub_temp_storage.data()), max_cub_storage_bytes, thrust::raw_pointer_cast(device_token_ids.data()), contiguous_token_ids, thrust::raw_pointer_cast(device_num_selected.data()), num_code_points, select_op);
  assertCudaSuccess(hipPeekAtLastError());  
  
  // Repurpose start word indices since it is the same size and type as the required output.
  uint32_t* token_id_counts = device_start_word_indices;
  device_start_word_indices = nullptr;
  hipcub::DeviceScan::InclusiveSum(thrust::raw_pointer_cast(cub_temp_storage.data()), max_cub_storage_bytes, thrust::raw_pointer_cast(device_tokens_per_word.data()), token_id_counts, num_code_points);
  assertCudaSuccess(hipPeekAtLastError());  

  constexpr uint16_t sen_update_num_threads = 64;       
  size_t SEN_KERNEL_BLOCKS = (num_sentences + sen_update_num_threads - 1) / sen_update_num_threads;                  
  update_sentence_lengths<<<SEN_KERNEL_BLOCKS, sen_update_num_threads>>>(device_sentence_offsets, token_id_counts, num_sentences);
  assertCudaSuccess(hipPeekAtLastError());  

  // Grab total number of token ids from the device
  uint32_t total_token_ids = 0;
  assertCudaSuccess(hipMemcpy(&total_token_ids, token_id_counts + num_code_points - 1, sizeof(total_token_ids), hipMemcpyDeviceToHost)); 
  
  cp_and_length.length = total_token_ids;
}



GpuWordPieceTokenizer::~GpuWordPieceTokenizer() {
}
